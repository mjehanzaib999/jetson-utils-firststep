#include "hip/hip_runtime.h"
#include "cudaBrightness.h"

template<typename T>
__global__ void increaseBrightness(T* image, int width, int height, int channels, float brightness_factor) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int index = (y * width + x) * channels;

        // Add brightness to each pixel component (R, G, B)
        for (int c = 0; c < channels; ++c) {
            int newValue = image[index + c] * brightness_factor;
            // Ensure the pixel value stays qwithin the range [0, 255]
            image[index + c] = min(max(newValue, 0), 255);
        }
    }
}


// launchResize
template<typename T>
static hipError_t launchBrightness( T* input, size_t input_Width, size_t input_Height, int width, int height, int channels, float brightness_factor)
{
	if( !input )
		return hipErrorInvalidDevicePointer;

	if( input_Width == 0 || input_Height == 0 )
		return hipErrorInvalidValue;

    // Calculate grid and block sizes
    const dim3 blockDim(8, 8);
    dim3 gridDim(iDivUp(input_Width,blockDim.x), iDivUp(input_Height,blockDim.y));

    // Launch the kernel
    increaseBrightness<T><<<gridDim, blockDim>>>(input, width, height, channels, brightness_factor);


	return CUDA(hipGetLastError());
}

// cudaResize (uint8 grayscale)
hipError_t cudaBrightness( uint8_t* input, size_t input_Width, size_t input_Height, int width, int height, int channels, float brightness_factor)
{
	return launchBrightness<uint8_t>(input, input_Width, input_Height, width, height, channels, brightness_factor);
}

// // cudaResize (uchar3 grayscale)
// hipError_t cudaBrightness( uchar3* input, int width, int height, int channels, float brightness_factor)
// {
// 	return launchBrightness<uchar3>(input, width, height, channels, brightness_factor);
// }

// // cudaResize (uchar3 grayscale)
// hipError_t cudaBrightness( uchar4* input, int width, int height, int channels, float brightness_factor)
// {
// 	return launchBrightness<uchar4>(input, width, height, channels, brightness_factor);
// }

// // cudaResize (uchar3 grayscale)
// hipError_t cudaBrightness( float3* input, int width, int height, int channels, float brightness_factor)
// {
// 	return launchBrightness<float3>(input, width, height, channels, brightness_factor);
// }

// // cudaResize (uchar3 grayscale)
// hipError_t cudaBrightness( float4* input, int width, int height, int channels, float brightness_factor)
// {
// 	return launchBrightness<float4>(input, width, height, channels, brightness_factor);
// }


// cudaResize (float grayscale)
hipError_t cudaBrightness( float* input, size_t input_Width, size_t input_Height, int width, int height, int channels, float brightness_factor)
{
	return launchBrightness<float>(input, width,input_Width, input_Height,  height, channels, brightness_factor);
}

//-----------------------------------------------------------------------------------
hipError_t cudaBrightness( void* input, size_t input_Width, size_t input_Height, int width, int height, int channels, float brightness_factor,
				    imageFormat format)
{
	if( format == IMAGE_RGB8 || format == IMAGE_BGR8 )
        return cudaBrightness((uint8_t*)input, input_Width, input_Height, width, height, channels, brightness_factor);
		//return cudaBrightness((uchar3*)input, width, height, channels, brightness_factor);
	// else if( format == IMAGE_RGBA8 || format == IMAGE_BGRA8 )
	// 	return cudaBrightness((uchar4*)input, width, height, channels, brightness_factor);
	else if( format == IMAGE_RGB32F || format == IMAGE_BGR32F )
		return cudaBrightness((float*)input,input_Width, input_Height,  width, height, channels, brightness_factor);
	else if( format == IMAGE_RGBA32F || format == IMAGE_BGRA32F )
		return cudaBrightness((float*)input,input_Width, input_Height,  width, height, channels, brightness_factor);
	else if( format == IMAGE_GRAY8 )
		return cudaBrightness((uint8_t*)input,input_Width, input_Height,  width, height, channels, brightness_factor);
	else if( format == IMAGE_GRAY32F )
		return cudaBrightness((float*)input, input_Width, input_Height, width, height, channels, brightness_factor);

	LogError(LOG_CUDA "cudaBrightness() -- invalid image format '%s'\n", imageFormatToStr(format));
	LogError(LOG_CUDA "                supported formats are:\n");
	LogError(LOG_CUDA "                    * gray8\n");
	LogError(LOG_CUDA "                    * gray32f\n");
	LogError(LOG_CUDA "                    * rgb8, bgr8\n");
	LogError(LOG_CUDA "                    * rgba8, bgra8\n");
	LogError(LOG_CUDA "                    * rgb32f, bgr32f\n");
	LogError(LOG_CUDA "                    * rgba32f, bgra32f\n");

	return hipErrorInvalidValue;
}